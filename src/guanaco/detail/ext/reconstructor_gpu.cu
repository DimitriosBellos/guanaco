#include "hip/hip_runtime.h"
/*
 * This file is part of guanaco-ctf.
 * Copyright 2021 Diamond Light Source
 * Copyright 2021 Rosalind Franklin Institute
 *
 * Author: James Parkhurst
 *
 * guanaco-ctf is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * guanaco-ctf is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with guanaco-ctf. If not, see <http:// www.gnu.org/licenses/>.
 */
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <thrust/complex.h>
#include <thrust/device_vector.h>

#include <guanaco/guanaco.h>

#define GUANACO_ASSERT_CUDA(assertion)                                   \
  if (!assertion) {                                                      \
    auto error = hipGetLastError();                                     \
    throw guanaco::Error(__FILE__, __LINE__, hipGetErrorString(error)); \
  }

namespace guanaco {

/* template <> */
/* class FFT<e_device> { */
/* public: */
/*   using size_type = std::size_t; */

/*   FFT(size_type size, size_type nbatch) : plan_forward_(0), plan_inverse_(0) { */
/*     create_plan(size, nbatch); */
/*   } */

/*   ~FFT() { */
/*     destroy_plan(); */
/*   } */

/*   void forward(void *in, void *out) const { */
/*     auto i = reinterpret_cast<hipfftReal *>(in); */
/*     auto o = reinterpret_cast<hipfftComplex *>(out); */
/*     hipfftExecR2C(plan_forward_, i, o); */
/*   } */

/*   void inverse(void *in, void *out) const { */
/*     auto i = reinterpret_cast<hipfftComplex *>(in); */
/*     auto o = reinterpret_cast<hipfftReal *>(out); */
/*     hipfftExecC2R(plan_inverse_, i, o); */
/*   } */

/* protected: */
/*   void create_plan(size_type size, size_type nbatch) { */
/*     hipfftPlan1d(&plan_forward_, size, HIPFFT_R2C, nbatch); */
/*     hipfftPlan1d(&plan_inverse_, size, HIPFFT_C2R, nbatch); */
/*   } */
  
/*   /1* void create_plan(size_type xsize, size_type ysize, size_type nbatch) { *1/ */
/*   /1*   hipfftPlan2d(&plan_forward_, ysize, xsize, HIPFFT_R2C); *1/ */
/*   /1*   hipfftPlan2d(&plan_inverse_, ysize, xsize, HIPFFT_C2R); *1/ */
/*   /1* } *1/ */

/*   void destroy_plan() { */
/*     hipDeviceSynchronize(); */
/*     hipfftDestroy(plan_forward_); */
/*     hipfftDestroy(plan_inverse_); */
/*   } */

/*   hipfftHandle plan_forward_; */
/*   hipfftHandle plan_inverse_; */
/* }; */

template <>
class Filter<e_device> {
public:
  using size_type = std::size_t;

  using vector_type = std::vector<float>;

  Filter(size_type num_pixels, size_type num_angles, size_type num_defocus);

  void operator()(float *data) const;

  const vector_type &filter() const;

protected:
  vector_type create_filter(size_type size) const;

  size_type num_pixels_;
  size_type num_angles_;
  size_type num_defocus_;
  vector_type filter_;
  FFT<e_device> fft_;
};

Filter<e_device>::Filter(size_type num_pixels,
                         size_type num_angles,
                         size_type num_defocus)
    : num_pixels_(num_pixels),
      num_angles_(num_angles),
      num_defocus_(num_defocus),
      filter_(create_filter(num_pixels_ + 1)),
      fft_(FFT<e_device>::make_1d_batch(num_pixels_ * 2, num_angles_)) {
  GUANACO_ASSERT(num_pixels_ > 0);
  GUANACO_ASSERT(num_angles_ > 0);
  GUANACO_ASSERT(num_defocus_ > 0);
}

const Filter<e_device>::vector_type &Filter<e_device>::filter() const {
  return filter_;
}

Filter<e_device>::vector_type Filter<e_device>::create_filter(size_type size) const {
  GUANACO_ASSERT(size > 0);

  // Initialise the filter array
  auto filter = vector_type(size, 0);

  // Create a Ramp filter
  for (auto i = 0; i < size; ++i) {
    float w = float(i) / float(size - 1);
    filter[i] = w;
  }

  // Return the filter
  return filter;
}

void Filter<e_device>::operator()(float *data) const {
  GUANACO_ASSERT(filter_.size() == num_pixels_ + 1);

  // Make some typedefs
  using device_vector_f = thrust::device_vector<float>;
  using device_vector_c = thrust::device_vector<thrust::complex<float>>;

  // Copy the filter to the device
  auto filter_d = device_vector_c(filter_.size(), 0);
  thrust::copy(filter_.begin(), filter_.end(), filter_d.begin());

  // When taking the FT of the data, we are going from real to complex so the
  // output array only stores the non-redundant complex coefficients so the
  // complex array requires (N/2 + 1) elements.
  for (auto j = 0; j < num_defocus_; ++j) {
    auto rows_c = device_vector_c(num_angles_ * filter_.size(), 0);
    auto rows_f = device_vector_f(num_angles_ * num_pixels_ * 2, 0);

    // Get a pointer to the sinogram
    auto data_ptr = data + j * num_angles_*num_pixels_;

    // Copy the rows of the sinogram to a zero padded array. 
    for (auto i = 0; i < num_angles_; ++i) {
      thrust::copy(data_ptr + i * num_pixels_,
                   data_ptr + i * num_pixels_ + num_pixels_,
                   rows_f.begin() + i * num_pixels_ * 2);
    }

    // Take the FT of the rows of the data
    fft_.forward(rows_f.data().get(), rows_c.data().get());

    // Apply the filter to each projection
    for (auto i = 0; i < num_angles_; ++i) {
      thrust::transform(filter_d.begin(),
                        filter_d.end(),
                        rows_c.begin() + i * filter_.size(),
                        rows_c.begin() + i * filter_.size(),
                        thrust::multiplies<thrust::complex<float>>());
    }

    // Take the inverse FT of the rows of the data
    fft_.inverse(rows_c.data().get(), rows_f.data().get());

    // Scale the filtered data
    auto factor = num_pixels_ * 2;
    thrust::transform(
      rows_f.begin(), rows_f.end(), rows_f.begin(), [factor] __device__(auto x) {
        return x / factor;
      });

    // Copy the filtered data back into the array
    for (int i = 0; i < num_angles_; ++i) {
      thrust::copy(rows_f.begin() + i * num_pixels_ * 2,
                   rows_f.begin() + i * num_pixels_ * 2 + num_pixels_,
                   data_ptr + i * num_pixels_);
    }
  }
}

namespace detail {

  // Declare global variables
  namespace global {

    const size_t MAX_ANGLES = 4096;

    typedef texture<float, 3, hipReadModeElementType> texture_type;

    static texture_type sinogram;

    __constant__ float angle_param_b[MAX_ANGLES];
    __constant__ float angle_param_a[MAX_ANGLES];
    __constant__ float angle_offset[MAX_ANGLES];
    __constant__ float angle_scale[MAX_ANGLES];

  }  // namespace global

  namespace g = global;

  /**
   * A functor that implements the back projection per voxel
   */
  struct BPFunction {

    size_t num_angles;
    size_t grid_width;
    size_t grid_height;
    float output_scale;
    float dscale;
    float doffset;

    BPFunction(size_t num_angles_,
               size_t grid_width_,
               size_t grid_height_,
               float output_scale_,
               float dscale_,
               float doffset_)
        : num_angles(num_angles_),
          grid_width(grid_width_),
          grid_height(grid_height_),
          output_scale(output_scale_),
          dscale(dscale_),
          doffset(doffset_) {
      GUANACO_ASSERT(num_angles_ <= g::MAX_ANGLES);
    }

    __device__ float operator()(size_t index, float voxel) const {
      // Get the X and Y indices
      const size_t j = index / grid_width;
      const size_t i = index - j * grid_width;

      // Compute the x and y coordinates
      const float x = (i - 0.5 * grid_width + 0.5);
      const float y = (j - 0.5 * grid_height + 0.5);

      // Loop through all the angles and compute the value of the voxel
      float value = 0.0f;
      for (size_t angle = 0; angle < num_angles; ++angle) {
        // Get parameters
        const float a = g::angle_param_a[angle];
        const float b = g::angle_param_b[angle];
        const float c = g::angle_offset[angle];
        const float scale = g::angle_scale[angle];

        // Compute the pixel and defocus coordinate
        const float pixel = a * x + b * y + c;
        const float height = b * x - a * y;
        const float defocus = height * dscale + doffset;

        // Sum the sinogram value for the pixel and angle
        value += tex3D(g::sinogram, pixel, angle + 0.5, defocus) * scale;
      }

      // Add the contribution to the voxel
      return voxel + value * output_scale;
    }
  };

  struct BP {
    using size_type = std::size_t;

    hipArray *sinogram_array_;
    size_type num_pixels_;
    size_type num_angles_;
    size_type num_defocus_;
    float pixel_size_;
    float min_defocus_;
    float max_defocus_;

    BP(size_type num_pixels,
       size_type num_angles,
       size_type num_defocus,
       float centre,
       float pixel_size,
       float min_defocus,
       float max_defocus,
       const float *sinogram,
       const float *angles)
        : sinogram_array_(nullptr), 
          num_pixels_(num_pixels), 
          num_angles_(num_angles), 
          num_defocus_(num_defocus), 
          pixel_size_(pixel_size), 
          min_defocus_(min_defocus), 
          max_defocus_(max_defocus) {

      // Check input
      GUANACO_ASSERT(num_pixels_ > 0);
      GUANACO_ASSERT(num_angles_ > 0);
      GUANACO_ASSERT(num_defocus_ > 0);
      GUANACO_ASSERT(pixel_size_ > 0);
      GUANACO_ASSERT(max_defocus_ >= min_defocus_);

      // Copy the angle data to device symbols
      copy_angles(angles, num_angles_, centre);

      // Copy the sinogram to the texture memory
      copy_sinogram(sinogram, num_pixels, num_angles, num_defocus);
    }

    ~BP() {
      hipFreeArray(sinogram_array_);
    }

    void copy_angles(const float *angles, size_type num_angles, float centre) const {
      // Copy the data to the symbol. For some reason I can't pass the symbol
      // pointer as normal (no idea) so I have to pass a pointer to the
      // symbol array pointer and then dereference!
      auto copy = [](auto symbol, auto data, auto n) {
        GUANACO_ASSERT(n <= g::MAX_ANGLES);
        auto error = hipMemcpyToSymbol(HIP_SYMBOL(
          *symbol), data, n * sizeof(float), 0, hipMemcpyHostToDevice);
        GUANACO_ASSERT_CUDA(error == hipSuccess);
      };

      // Allocate host vectors
      auto angle_param_b = thrust::host_vector<float>(num_angles);
      auto angle_param_a = thrust::host_vector<float>(num_angles);
      auto angle_offset = thrust::host_vector<float>(num_angles);
      auto angle_scale = thrust::host_vector<float>(num_angles);

      // Compute the quanities to store in the symbols
      for (auto i = 0; i < num_angles; ++i) {
        // Get the ray vector and length of the ray vector
        auto angle = angles[i];
        auto dir_x = std::cos(angle);
        auto dir_y = std::sin(angle);
        auto det_x0 = -centre * dir_x;
        auto det_y0 = -centre * dir_y;
        auto ray_length = 1.0;  // std::sqrt(dir_x * dir_x + (-dir_y) * (-dir_y));
        auto d = 1.0;           // dir_x * dir_x - dir_y * (-dir_y);

        // Fill the arrays
        angle_param_a[i] = dir_x / d;
        angle_param_b[i] = (-dir_y) / d;
        angle_offset[i] = (det_y0 * (-dir_y) - det_x0 * dir_x) / d;
        angle_scale[i] = ray_length / std::abs(d);
      }

      // Copy the arrays to the symbols
      copy(&g::angle_param_b, angle_param_b.data(), num_angles);
      copy(&g::angle_param_a, angle_param_a.data(), num_angles);
      copy(&g::angle_offset, angle_offset.data(), num_angles);
      copy(&g::angle_scale, angle_scale.data(), num_angles);
    }

    void copy_sinogram(const float *sinogram,
                       size_type num_pixels,
                       size_type num_angles,
                       size_type num_defocus) {
      // Allocate a cuda array needed to bind 3D texture
      auto channel_desc = hipCreateChannelDesc<float>();
      auto extent = make_hipExtent(num_pixels, num_angles, num_defocus);
      auto error = hipMalloc3DArray(&sinogram_array_, &channel_desc, extent);
      GUANACO_ASSERT_CUDA(error == hipSuccess);

      // Copy the data
      hipMemcpy3DParms copy_params{0};
      copy_params.srcPtr = make_hipPitchedPtr(
        (void *)sinogram, extent.width * sizeof(float), extent.width, extent.height);
      copy_params.dstArray = sinogram_array_;
      copy_params.extent = extent;
      copy_params.kind = hipMemcpyDeviceToDevice;
      hipMemcpy3D(&copy_params);

      // Set texture parameters.
      // For examples and pixels outside the expected range, this sets the
      // value to zero (border). For defocus outside of expected range, use the
      // closest (clamp).
      g::sinogram.addressMode[0] = hipAddressModeBorder;
      g::sinogram.addressMode[1] = hipAddressModeBorder;
      g::sinogram.addressMode[2] = hipAddressModeClamp;
      g::sinogram.filterMode = hipFilterModeLinear;
      g::sinogram.normalized = false;

      // Bind the texture to the array
      error = hipBindTextureToArray(g::sinogram, sinogram_array_, channel_desc);
      GUANACO_ASSERT_CUDA(error == hipSuccess);
    }

    void launch(float *reconstruction,
                size_type grid_width,
                size_type grid_height,
                float scale) const {
      // Check the input
      GUANACO_ASSERT(num_defocus_ == 1 || max_defocus_ > min_defocus_);

      // Compute the defocus scale and offset
      auto dscale = num_defocus_ > 1 
        ? num_defocus_ * pixel_size_ / (max_defocus_ - min_defocus_)
        : 0;
      auto doffset = -dscale * (min_defocus_ / pixel_size_);

      // Get some other quantities
      auto grid_size = grid_width * grid_height;
      auto index = thrust::counting_iterator<size_t>(0);
      auto recon = thrust::device_pointer_cast(reconstruction);

      // Initialise the functor
      BPFunction func(num_angles_, grid_width, grid_height, scale, dscale, doffset);

      // Do the reconstruction
      thrust::transform(index, index + grid_size, recon, recon, func);
    }
  };

}  // namespace detail

Reconstructor_t<e_device>::Reconstructor_t(const Config &config) : config_(config) {
  GUANACO_ASSERT(config_.device == e_device);
  GUANACO_ASSERT(config_.is_valid());
}

void Reconstructor_t<e_device>::operator()(const float *sinogram,
                                           float *reconstruction) const {
  Filter<e_device> filter_(config_.num_pixels, config_.num_angles, config_.num_defocus);

  // A function to set the gpu index
  auto set_gpu_index = [](int index) {
    if (index >= 0) {
      hipSetDevice(index);
      auto error = hipGetLastError();
      GUANACO_ASSERT_CUDA((error == hipSuccess)
                          || (error == hipErrorSetOnActiveProcess));
    }
  };

  // Make some typedefs
  using vector_type = thrust::device_vector<float>;

  // Get the sinogram and reconstruction sizes along with the number of
  // angles and the pixel area
  auto sino_size = config_.sino_size();
  auto grid_size = config_.grid_size();

  // Allocate device vectors for sinogram and reconstruction
  auto sinogram_d = vector_type(sinogram, sinogram + sino_size);
  auto reconstruction_d = vector_type(grid_size, 0);

  // Set the gpu
  set_gpu_index(config_.gpu_index);

  // Filter the sinogram
  filter_(sinogram_d.data().get());

  // Perform the backprojection
  project(sinogram_d.data().get(), reconstruction_d.data().get());

  // Copy the data back to the host ptr
  thrust::copy(reconstruction_d.begin(), reconstruction_d.end(), reconstruction);
}

void Reconstructor_t<e_device>::project(const float *sinogram,
                                        float *reconstruction) const {

  // Check the dimensions against the maximum texture size
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  GUANACO_ASSERT(prop.maxTexture3D[1] >= detail::g::MAX_ANGLES);
  GUANACO_ASSERT(prop.maxTexture3D[0] >= config_.num_pixels);
  GUANACO_ASSERT(prop.maxTexture3D[1] >= config_.num_angles);
  GUANACO_ASSERT(prop.maxTexture3D[2] >= config_.num_defocus);

  // Compute the scale
  auto scale = M_PI / (2 * config_.num_angles);

  // Initialise the back projector class
  auto bp = detail::BP(config_.num_pixels,
                       config_.num_angles,
                       config_.num_defocus,
                       config_.centre,
                       config_.pixel_size,
                       config_.min_defocus,
                       config_.max_defocus,
                       sinogram,
                       config_.angles.data());

  // Launch the back projector
  bp.launch(reconstruction, config_.grid_width, config_.grid_height, scale);
}

}  // namespace guanaco
